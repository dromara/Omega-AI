#define BLOCK 1024 

#include <hip/hip_runtime.h>
 

__device__ void kernel_compute_statistics(
    float* scores,
    float* local_rowmax,
    float global_rowmax_old,
    float global_rowsum_old,
    float* global_rowmax_new,
    float* global_rowsum_new,
    int num_rows_per_tile,
    int thread_idx,
    int local_row_idx,
    int dimension)
{
    if(thread_idx % dimension == 0){ 
        //compute rowsums for S_ij
        float l_ij = 0.0f;
        float m_i_new = 0.0f;
        float l_i_new = 0.0f;
        for (int i = 0; i < num_rows_per_tile; i++){
            l_ij += scores[local_row_idx * num_rows_per_tile + i]; //l_ij doenst need to be written to SRAM
        }
        //compute new global rowmax statistics 
        m_i_new = fmax(global_rowmax_old, local_rowmax[local_row_idx]);
        global_rowmax_new[local_row_idx] = m_i_new; //Reuse the shared memory allocated to Q_i, since we dont use it anymore after computing S_ij

        //compute new global rowsum statistics
        l_i_new = expf(global_rowmax_old - m_i_new) * global_rowsum_old + expf(local_rowmax[local_row_idx] - m_i_new) * l_ij; 
        global_rowsum_new[local_row_idx] = l_i_new;  
    }
}

__device__ void kernel_reduction_max( //TODO: not actually reduction yet
    float* scores, 
    float* local_rowmax, 
    int num_rows_per_tile, 
    int dimension,  
    int thread_idx,
    int block_idx)
{
    if(thread_idx < num_rows_per_tile){ //S_ij is square, so num_rows_per_tile accounts for both row and column dimension
        float max_val = -INFINITY;
        for (int i = 0; i < num_rows_per_tile; i++){
            auto s_ij = scores[thread_idx * num_rows_per_tile + i];
            max_val = fmax(max_val, s_ij);
        }
        local_rowmax[thread_idx] = max_val; //Resue Q_i allocated memory
    }   
}

__device__ void inner_product_matmul(
    float* Q_i, 
    float* K_j, 
    float* scores, 
    int num_rows_per_block,
    int dimension, 
    int thread_idx, 
    int thread_idx_limit,
    float scaling_factor)
{
    if (thread_idx < thread_idx_limit){
        //each threads computes one output value
        float temp = 0.0f;
        int local_matrix_row_index = thread_idx / num_rows_per_block;
        for(int k = 0; k < dimension; k++){
            temp += Q_i[local_matrix_row_index * dimension + k] * K_j[(thread_idx % num_rows_per_block) * dimension + k]; //Q_i * K^T_j
        }
        scores[thread_idx] = scaling_factor * temp;
    }
}

__device__ float outer_product_matmul(
    float* scores,
    float* V_j, 
    int num_rows_per_block,
    int dimension,
    int thread_idx,
    int thread_idx_limit
    )
{
    if(thread_idx < thread_idx_limit){ //TODO: fix edge case for when last tile does not have same amount of rows
        float temp = 0.0f;
        for (int k = 0; k < num_rows_per_block; k++){
            temp += scores[(thread_idx / dimension) * num_rows_per_block + k] * V_j[k * dimension + (thread_idx % dimension)];
        }
        return temp;
    };
    return 0.0f;
}

__device__ float* shared_memory_proxy()
{
    extern __shared__ unsigned char memory[];
    return reinterpret_cast<float*>(memory);
}

extern "C"
__global__ void forward_attention_kernel(
    float* query,
    float* key,
    float* value,
    float* outputs,
    float* rowmax_statistics, 
    float* rowsum_statistics,
    int batch_size, int sequence_length, int dimension,
    int block_size,
    int num_rows_per_block,
    int num_blocks_per_sample)
{
    //SRAM
    extern __shared__ float sharedMemory[];
    //float* sharedMemory = shared_memory_proxy<float>();
    float* Q_i = &sharedMemory[0];
    float* K_j = Q_i + block_size;
    float* V_j = Q_i + 2*block_size; 
    float* scores = &sharedMemory[0]; //Reuse Q_i allocated SRAM space
    float* local_rowmax = scores + num_rows_per_block * num_rows_per_block; 
    float* global_rowmax_new = local_rowmax + num_rows_per_block; 
    float* global_rowsum_new = global_rowmax_new + num_rows_per_block; 
    
    //compute indexes
    int batch_idx = blockIdx.x; 
    int local_row_idx = threadIdx.x / dimension; 
    int col_idx = threadIdx.x % dimension; // global_col_idx == local_col_idx in this sense

    //scaling factor
    float scaling_factor = 1.0f / (sqrtf(static_cast<float>(dimension)));
	
	int onceLen = num_rows_per_block * dimension;
	
    if(batch_idx < batch_size && local_row_idx < num_rows_per_block){ 
        for(int j = 0; j < num_blocks_per_sample; j++){
            //Load K_j, V_j to SRAM
            
            int offset = batch_idx * onceLen + (j * num_rows_per_block + local_row_idx) * dimension + col_idx;
            
            K_j[threadIdx.x] = key[offset]; // K_j
            V_j[threadIdx.x] = value[offset]; // V_j - Not very coalessed for when we do our matmuls later.... 

            for(int i = 0; i < num_blocks_per_sample; i++){ //i gives us which tile we are on for Q along the row-axis

                int global_row_idx_i = i * num_rows_per_block + local_row_idx;
				
				int qidx = batch_idx * onceLen + global_row_idx_i * dimension + col_idx;
				
                //Load Q_i, m_i, l_i to SRAM - O_i is unecessary 
                Q_i[threadIdx.x] = query[qidx]; 
                
                //Compute attention scores Q_i*K^T_j
                __syncthreads(); //necessary because utilized threads all come from the first row in the tile, but some of them operate on values from other rows in the tile
                inner_product_matmul(Q_i, K_j, scores, num_rows_per_block, dimension, threadIdx.x, num_rows_per_block * num_rows_per_block, scaling_factor); 
                // __syncthreads(); 
 
                //compute statistics - brute force it for now...
                kernel_reduction_max(scores, local_rowmax, num_rows_per_block, dimension, threadIdx.x, blockIdx.x);
                __syncthreads();

                if(threadIdx.x < num_rows_per_block*num_rows_per_block){ 
                    scores[threadIdx.x] = expf(scores[threadIdx.x] - local_rowmax[threadIdx.x / num_rows_per_block]); //P_ij 
                }
                __syncthreads();

                float global_rowmax_old = rowmax_statistics[batch_idx * num_rows_per_block + global_row_idx_i];
                float global_rowsum_old = rowsum_statistics[batch_idx * num_rows_per_block + global_row_idx_i]; 
                kernel_compute_statistics(scores, local_rowmax, global_rowmax_old, global_rowsum_old, global_rowmax_new, global_rowsum_new, num_rows_per_block, threadIdx.x, local_row_idx, dimension); //pretty sure its good
                __syncthreads();
            
                //compute attention outputs (from here on out its all element-wise so we dont need to sync threads)
                auto m_i_new = global_rowmax_new[local_row_idx];

                float old_output_adjusted = (global_rowsum_old * expf(global_rowmax_old - m_i_new)) * outputs[qidx]; 
                float local_attention_adjusted = outer_product_matmul(scores, V_j, num_rows_per_block, dimension, threadIdx.x, num_rows_per_block * dimension); //TODO: num_rows_per_block*dimension doesnt account for edge_case of non-divisible total-rows
                local_attention_adjusted = expf(local_rowmax[local_row_idx] - m_i_new) * local_attention_adjusted; 

                //Write to global memory (HBM)
                outputs[qidx] = (1 / (global_rowsum_new[local_row_idx])) * (old_output_adjusted + local_attention_adjusted); 
                if(threadIdx.x < num_rows_per_block){
					int ridx = batch_idx * num_rows_per_block + (i * num_rows_per_block + threadIdx.x % num_rows_per_block);
                    rowmax_statistics[ridx] = global_rowmax_new[threadIdx.x];
                    rowsum_statistics[ridx] = global_rowsum_new[threadIdx.x]; 
                }
            }
        }
    }
}


