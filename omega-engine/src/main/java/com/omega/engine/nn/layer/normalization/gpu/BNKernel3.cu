
#include <hip/hip_runtime.h>
#define BLOCK 1024 

extern "C"
__global__ void  fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;
            local[id] += (i+id < spatial) ? x[index] : 0;
        }
    }

    __syncthreads();

    if(id == 0){
        mean[filter] = 0;
        for(i = 0; i < threads; ++i){
            mean[filter] += local[i];
        }
        mean[filter] /= spatial * batch;
    }
}

extern "C"
__global__ void  fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;

            local[id] += (i+id < spatial) ? powf((x[index] - mean[filter]), 2) : 0;
        }
    }

    __syncthreads();

    if(id == 0){
        variance[filter] = 0;
        for(i = 0; i < threads; ++i){
            variance[filter] += local[i];
        }
        variance[filter] /= (spatial * batch - 1);
        //variance[filter] /= (spatial * batch);
    }
}

extern "C"
__global__ void normalize_kernel(int N, float *x, float *z, float *out, float *mean, float *variance, float *gama, float *beta,int batch, int filters, int spatial,float eta)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;
    int f = (index/spatial)%filters;
    z[index] = (x[index] - mean[f])/(sqrtf(variance[f] + eta));
    out[index] = z[index] * gama[f] + beta[f];
}

extern "C"
__global__ void normalize_test_kernel(int N, float *x, float *out, float *mean, float *variance, float *gama, float *beta,int batch, int filters, int spatial,float eta)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;
    int f = (index/spatial)%filters;
    out[index] = (x[index] - mean[f])/(sqrtf(variance[f] + eta)) * gama[f] + beta[f];
}

extern "C"
__global__ void mwa_kernel(float* mean,float* var,float* runingMean,float* runingVar,int n,float momentum)
{
    
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < n; index += blockDim.x * gridDim.x) {
		
		runingMean[index] = (1.0f - momentum) * runingMean[index] + momentum * mean[index];
		
		runingVar[index] = (1.0f - momentum) * runingVar[index] + momentum * var[index];
		
	}

}

extern "C"
__global__ void backward_bias_conn_kernel(float *bias_updates, float *delta, int batch, int n)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= n) return;
    int b;
    float sum = 0;
    for(b = 0; b < batch; ++b){
        int i = b*n + index;
        sum += delta[i];
    }
    bias_updates[index] += sum;
}

extern "C"
__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int i,b;
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;
    for(b = 0; b < batch; ++b){
        for(i = 0; i < size; i += BLOCK){
            int index = p + i + size*(filter + n*b);
            sum += (p+i < size) ? delta[index] : 0;
        }
    }
    part[p] = sum;
    __syncthreads();
    if (p == 0) {
        for(i = 0; i < BLOCK; ++i) bias_updates[filter] += part[i];
    }
}

extern "C"
__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int i,b;
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;
    for(b = 0; b < batch; ++b){
        for(i = 0; i < size; i += BLOCK){
            int index = p + i + size*(filter + n*b);
            sum += (p+i < size) ? delta[index]*x_norm[index] : 0;
        }
    }
    part[p] = sum;
    __syncthreads();
    if (p == 0) {
        for(i = 0; i < BLOCK; ++i) scale_updates[filter] += part[i];
    }
}

extern "C"
__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    if(offset < size) output[(batch*n+filter)*size + offset] *= biases[filter];
}

extern "C"
__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;
            local[id] += (i+id < spatial) ? delta[index] : 0;
        }
    }

    __syncthreads();

    if(id == 0){
        mean_delta[filter] = 0;
        for(i = 0; i < threads; ++i){
            mean_delta[filter] += local[i];
        }
        mean_delta[filter] *= (-1.f/sqrtf(variance[filter] + .00001f));
    }
}

extern "C"
__global__ void  fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;

            local[id] += (i+id < spatial) ? delta[index]*(x[index] - mean[filter]) : 0;
        }
    }

    __syncthreads();

    if(id == 0){
        variance_delta[filter] = 0;
        for(i = 0; i < threads; ++i){
            variance_delta[filter] += local[i];
        }
        variance_delta[filter] *= -.5f * powf(variance[filter] + .00001f, -1.5f);
    }
}

extern "C"
__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;
    int f = (index/spatial)%filters;
    
    delta[index] = delta[index] * 1.0f/(sqrtf(variance[f] + .00001f)) + variance_delta[f] * 2.f * (x[index] - mean[f]) / (spatial * batch) + mean_delta[f]/(spatial*batch);
}