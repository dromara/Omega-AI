
#include <hip/hip_runtime.h>
#define BLOCK 1024 

extern "C"
__global__ void add_full_bias(float* output, float* biases, int N, int w)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= N) return;
    int j = i % w;
    output[i] += biases[j];
}

extern "C"
__global__ void add_bias(float* output, float* biases, int batch, int n, int size)
{
    
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= n*size*batch) return;
    int i = index % size;
    index /= size;
    int j = index % n;
    index /= n;
    int k = index;

    output[(k*n+j)*size + i] += biases[j];
	
}

extern "C"
__global__ void add_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    if(offset < size) output[(batch*n+filter)*size + offset] += biases[filter];
}

extern "C"
__global__ void backward_bias_conn_kernel(float *bias_updates, float *delta, int batch, int W)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= W) return;
    float sum = 0;
    for(int b = 0; b < batch; ++b){
        int i = b*W + index;
        sum += delta[i];
    }
    bias_updates[index] += sum;
}

extern "C"
__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int i,b;
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;
    for(b = 0; b < batch; ++b){
        for(i = 0; i < size; i += BLOCK){
            int index = p + i + size*(filter + n*b);
            sum += (p+i < size) ? delta[index] : 0;
        }
    }
    part[p] = sum;
    __syncthreads();
    if (p == 0) {
        for(i = 0; i < BLOCK; ++i) bias_updates[filter] += part[i];
    }
}