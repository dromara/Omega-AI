
#include <hip/hip_runtime.h>
#define BLOCK 1024

extern "C"
__global__ void loss(int N,float *input, float *label, float *output, float beta)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= N) return;
	float x = input[id] - label[id];
	float abs_x = abs(x);
	if(abs_x < beta){
		output[id] = 0.5f * x * x / beta;
	}else{
		output[id] = abs_x - 0.5f * beta;
	}
}

extern "C"
__global__ void loss_back(int N,float *input, float *label, float *diff, float beta)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
  	if (id >= N) return;
  	float x = input[id] - label[id];
	float abs_x = abs(x);
	float delta = 1.0f / N;
	if(abs_x < beta){
		diff[id] = x * delta / beta;
	}else{
		if(x > 0){
			diff[id] = 1 * delta;
		}else if(x == 0){
			diff[id] = 0;
		}else{
			diff[id] = -1 * delta;
		}
	}
}
