
#include <hip/hip_runtime.h>
#define BLOCK 1024
#define FLT_MAX 3.402823466e+38F

extern "C"
__global__ void fill_kernel(int N, float ALPHA, float *X)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) X[i] = ALPHA;
}

extern "C"
__global__ void copy_kernel(int N,  float *X, int OFFX, float *Y, int OFFY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N){
    	Y[i + OFFY] = X[i + OFFX];
    }
}

extern "C"
__global__ void axpy_kernel(int N,  float *X, int OFFX, float *Y, int OFFY)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N){
    	Y[i + OFFY] += X[i + OFFX];
    }
}

extern "C"
__global__ void copy_number_kernel(int N,  float *X, float *Y, int n,int c,int h,int w,int start,int cp)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N){
    	int size = c * h * w;
    	int tn = i / size + start;
		int tc = i / h / w % c;
		int th = i / w % h;
		int tw = i % w;
		int index = tn * size + tc * h * w + th * w + tw;
		if(cp == 0){
			Y[i] = X[index];
		}else{
			X[index] = Y[i];
		}
    }
}

extern "C"
__global__ void copy_channel_kernel(int N,  float *X, float *Y, int n,int c,int h,int w,int start,int cp)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N){
    	int bc = N / n / h / w;
		int size = bc * h * w;
    	int tn = i / size;
		int tc = (i / h / w) % bc + start;
		int th = i / w % h;
		int tw = i % w;
		int index = tn * c * h * w + tc * h * w + th * w + tw;
    	if(cp == 0){
			Y[i] = X[index];
		}else{
			X[index] = Y[i];
		}
    }
}

extern "C"
__global__ void broadcast_kernel(int N, float *X, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i] = X[0];
}

extern "C"
__global__ void broadcast_number_kernel(int N, float *X, float *Y,int C,int H,int W)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
    	int n = i / C / H / W;
    	Y[i] = X[n];
    }
}

extern "C"
__global__ void broadcast_plus_kernel(int N, float *X, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i] += X[0];
}

extern "C"
__global__ void broadcast_number_plus_kernel(int N, float *X, float *Y,int C,int H,int W)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
    	int n = i / C / H / W;
    	Y[i] += X[n];
    }
}

extern "C"
__global__ void add_kernel(int N, float *X, float *Y, float *R)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) R[i] = X[i] + Y[i];
}

extern "C"
__global__ void add_axis_kernel(int N, float *X, float *Y, float *R,int axis)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N){
    	int yi = i / axis;
    	R[i] = X[i] + Y[yi];
    } 
}

extern "C"
__global__ void sum_kernel(int N, float *X, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < 1) {
	    for(int index = 0;index<N;index++){
	    	Y[0] += X[index];
	    }
    }
}

extern "C"
__global__ void sum_channel_kernel(int N, float *X, float *Y,int C,int H,int W)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
    	for(int index = 0;index<C * H * W;index++){
    		Y[i] += X[i * C * H * W + index];
    	}
    }
}

extern "C"
__global__ void sum_height_kernel(int N, float *X, float *Y,int C,int H,int W)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
    	Y[i] = 0;
    	for(int index = 0;index<H * W;index++){
    		Y[i] += X[i * H * W + index];
    	}
    }
}

extern "C"
__global__ void max_kernel(int N, float *X, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < 1) {
    	float max = -FLT_MAX;
	    for(int index = 0;index<N;index++){
	    	if(max <= X[index]){
	    		max = X[index];
	    	}
	    }
	    Y[0] = max;
    }
}

extern "C"
__global__ void max_channel_kernel(int N, float *X, float *Y,int C,int H,int W)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
    	float max = -FLT_MAX;
    	for(int index = 0;index<C * H * W;index++){
    		if(max <= X[i * C * H * W + index]){
	    		max = X[i * C * H * W + index];
	    	}
    	}
    	Y[i] = max;
    }
}

extern "C"
__global__ void max_backward_kernel(int N, float *D, float *X, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < 1) {
    	float max = -FLT_MAX;
    	int max_idx = 0;
	    for(int index = 0;index<N;index++){
	    	if(max <= X[index]){
	    		max = X[index];
	    		max_idx = index;
	    	}
	    }
	    Y[max_idx] += D[0];
    }
}

extern "C"
__global__ void max_channel_backward_kernel(int N, float *D, float *X, float *Y, int C, int H, int W)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
    	float max = -FLT_MAX;
    	int max_idx = 0;
    	for(int index = 0;index<C * H * W;index++){
    		if(max <= X[i * C * H * W + index]){
	    		max = X[i * C * H * W + index];
	    		max_idx = i * C * H * W + index;
	    	}
    	}
    	Y[max_idx] += D[i];
    }
}

extern "C"
__global__ void add_scalar_kernel(int N, float *X, float ALPHA, float *R)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) R[i] = X[i] + ALPHA;
}

extern "C"
__global__ void add_number_kernel(int N,  float *X, float *Y, int n,int c,int h,int w,int start)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N){
    	int size = c * h * w;
    	int tn = i / size + start;
		int tc = (i / h / w) % c;
		int th = i / w % h;
		int tw = i % w;
		int index = tn * size + tc * h * w + th * w + tw;
    	X[index] += Y[i];
    }
}

extern "C"
__global__ void add_channel_kernel(int N,  float *X, float *Y, int n,int c,int h,int w,int start)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N){
    	int bc = N / n / h / w;
		int size = bc * h * w;
    	int tn = i / size;
		int tc = (i / h / w) % bc + start;
		int th = i / w % h;
		int tw = i % w;
		int index = tn * c * h * w + tc * h * w + th * w + tw;
    	X[index] += Y[i];
    }
}

extern "C"
__global__ void sub_kernel(int N, float *X, float *Y, float *R)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) R[i] = X[i] - Y[i];
}

extern "C"
__global__ void sub_axis_kernel(int N, float *X, float *Y, float *R,int axis)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N){
    	int yi = i / axis;
    	R[i] = X[i] - Y[yi];
    } 
}

extern "C"
__global__ void sub_scalar_kernel(int N, float *X, float ALPHA, float *R)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) R[i] = X[i] - ALPHA;
}

extern "C"
__global__ void scalar_sub_kernel(int N, float ALPHA, float *X, float *R)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) R[i] = ALPHA - X[i];
}

extern "C"
__global__ void bool_kernel(int N, float *X, float *Y, float *R,float val)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N){
    	if(Y[i] == 1){
    		R[i] = val;
    	}else{
    		R[i] = X[i];
    	}
    } 
}

extern "C"
__global__ void mul_kernel(int N, float *X, float *Y, float *R)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) R[i] = X[i] * Y[i];
}

extern "C"
__global__ void mul_scalar_kernel(int N, float *X, float ALPHA, float *R)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) R[i] = X[i] * ALPHA;
}

extern "C"
__global__ void mul_plus_kernel(int N, float *X, float *Y, float *R)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) R[i] += X[i] * Y[i];
}

extern "C"
__global__ void mul_plus_scalar_kernel(int N, float *X, float ALPHA, float *R)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) R[i] += X[i] * ALPHA;
}

extern "C"
__global__ void mul_plus_scalar_axis_kernel(int N, float *X, float ALPHA, float *R, int axis)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N){
    	for(int xi = 0;xi<axis;xi++){
    		R[i] += X[i * axis + xi] * ALPHA;
    	}
    }
}

extern "C"
__global__ void div_kernel(int N, float *X, float *Y, float *R)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) R[i] = X[i] / Y[i];
}

extern "C"
__global__ void div_axis_kernel(int N, float *X, float *Y, float *R,int axis)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N){
    	int yi = i / axis;
    	R[i] = X[i] / Y[yi];
    } 
}

extern "C"
__global__ void div_scalar_kernel(int N, float *X, float ALPHA, float *R)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) R[i] = X[i] / ALPHA;
}

extern "C"
__global__ void scalar_div_kernel(int N, float *X, float ALPHA, float *R)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) R[i] = ALPHA / X[i];
}

extern "C"
__global__ void div_bGrad_kernel(int N, float *D, float *A, float *B, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i] += - 1.0f * D[i] * A[i] / (B[i] * B[i]); 
}

extern "C"
__global__ void div_bGrad_axis_kernel(int N, float *D, float *A, float *B, float *Y,int axis)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N){
    	for(int di = 0;di<axis;di++){
    		Y[i] += (- 1.0f * D[i * axis + di] * A[i * axis + di]) / (B[i] * B[i]); 
    	}
    } 
}

extern "C"
__global__ void div_scalar_bGrad_kernel(int N, float *D, float A, float *B, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i] += - 1.0f * D[i] * A / (B[i] * B[i]); 
}

extern "C"
__global__ void div_plus_kernel(int N, float *X, float *Y, float *R)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) R[i] += X[i] / Y[i];
}

extern "C"
__global__ void div_plus_axis_kernel(int N, float *X, float *Y, float *R, int axis)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N){
    	int yi = i / axis;
    	R[i] += X[i] / Y[yi];
    } 
}

extern "C"
__global__ void div_plus_scalar_kernel(int N, float *X, float ALPHA, float *R)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) R[i] += X[i] / ALPHA;
}

extern "C"
__global__ void scalar_plus_div_kernel(int N, float *X, float ALPHA, float *R)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) R[i] += ALPHA / X[i];
}

extern "C"
__global__ void pow_kernel(int N, float *X, float ALPHA, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i] = powf(X[i], ALPHA);
}

extern "C"
__global__ void sqrt_kernel(int N, float *X, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i] = sqrtf(X[i]);
}

extern "C"
__global__ void log_kernel(int N, float *X, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
    	//if(X[i] == 0){
    		//X[i] = 0.00000000000000000000001f;
    	//}
   	 	Y[i] = logf(X[i]);
    }
}

extern "C"
__global__ void exp_kernel(int N, float *X, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i] = expf(X[i]);
}

extern "C"
__global__ void sin_kernel(int N, float *X, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i] = sin(X[i]);
}

extern "C"
__global__ void cos_kernel(int N, float *X, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i] = cos(X[i]);
}

extern "C"
__global__ void tan_kernel(int N, float *X, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i] = tan(X[i]);
}

extern "C"
__global__ void tan_back_kernel(int N, float *X, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i] = 1 / powf(cos(X[i]), 2);
}

extern "C"
__global__ void atan_kernel(int N, float *X, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i] = atan(X[i]);
}

extern "C"
__global__ void atan_back_kernel(int N, float *X, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) Y[i] = 1.0f / (1 + X[i] * X[i]);
}

extern "C"
__global__ void clamp_kernel(int N, float *X, float min, float max, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
    	float val = X[i];
		if(val < min) {
			Y[i] = min;
		}else if(val > max) {
			Y[i] = max;
		}else {
			Y[i] = val;
		}
    }
}

extern "C"
__global__ void clamp_back_kernel(int N, float *X, float min, float max, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
    	float val = X[i];
		if(val < min || val > max) {
			Y[i] = 0;
		}else {
			Y[i] = 1;
		}
    }
}

extern "C"
__global__ void maximum_kernel(int N, float *X, float *Z, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
		if(X[i] >= Z[i]) {
			Y[i] = X[i];
		}else {
			Y[i] = Z[i];
		}
    }
}

extern "C"
__global__ void minimum_kernel(int N, float *X, float *Z, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
		if(X[i] < Z[i]) {
			Y[i] = X[i];
		}else {
			Y[i] = Z[i];
		}
    }
}

extern "C"
__global__ void maximum_back_kernel(int N, float *X, float *Z, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
		if(X[i] >= Z[i]) {
			Y[i] = 1;
		}else {
			Y[i] = 0;
		}
    }
}

extern "C"
__global__ void minimum_back_kernel(int N, float *X, float *Z, float *Y)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < N) {
		if(X[i] < Z[i]) {
			Y[i] = 1;
		}else {
			Y[i] = 0;
		}
    }
}

extern "C"
__global__ void transpose_kernel(int N, float *A, float *B,int m,int n)
{	
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	int r = i / n;
	int c = i % n;
    if (i < N)
    {
        B[c * m + r] = A[r * n + c];
    }
}

extern "C"
__global__ void permute_kernel(int N, float *data_in, float *data_out, int *perms, int *strides_in, int *strides_out, int NUM_AXES) {
    //int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (tid < N) {
        int offset_out = tid;
        int offset_tmp = offset_out;
        int offset_in = 0;
        for (int i = 0; i < NUM_AXES; i++) {
            offset_in += (offset_tmp / strides_out[i]) * strides_in[perms[i]];
            offset_tmp %= strides_out[i];
        }
        data_out[offset_out] = data_in[offset_in];
    }
}
