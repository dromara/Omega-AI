#include "hip/hip_runtime.h"
#define BLOCK 1024 
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

__host__ __device__ float ceil_div(float dividend, float divisor) {
    return (dividend + divisor-1) / divisor;
}

extern "C"
__global__ void groupnorm_forward_kernel(
    const float* x, const float* weight, const float* bias,
    float* out, float* mean, float* rstd,
    int B, int C, int img_size, int group_size, int n_groups
) {
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    __shared__ float shared_sum[32]; // block_size max is 1024 = 32 * 32 warps
    __shared__ float shared_sum2[32]; // warps will be writing into shared memeory after warp-reduce
    int num_warps = blockDim.x / 32;
    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;
    int block_pixels = img_size * group_size;
    // group index
    int g = blockIdx.x % n_groups;

    // move pointers
    x += blockIdx.x * img_size * group_size;
    out += blockIdx.x * img_size * group_size;
    // each block will only every acces group_size channels
    weight += g * group_size;
    bias += g * group_size;

    float thread_sum = 0.0f;
    float thread_sum2 = 0.0f;
    for (int i = threadIdx.x; i < block_pixels; i += blockDim.x) {
        float val = x[i];
        thread_sum += val;
        thread_sum2 += val * val;
    }

    // warp reduce
    float warp_sum = cg::reduce(warp, thread_sum, cg::plus<float>{});
    float warp_sum2 = cg:: reduce(warp, thread_sum2, cg::plus<float>{});
    // store warp sum into shared memory
    shared_sum[warp_id] = warp_sum;
    shared_sum2[warp_id] = warp_sum2;
    __syncthreads();
    
    // load warp sums from shared memory
    warp_sum = (lane_id < num_warps) ? shared_sum[lane_id] : 0.0f;
    warp_sum2 = (lane_id < num_warps) ? shared_sum2[lane_id] : 0.0f;
    float block_sum = cg::reduce(warp, warp_sum, cg::plus<float>{});
    float block_sum2 = cg::reduce(warp, warp_sum2, cg::plus<float>{});
    block_sum /= block_pixels;
    block_sum2 /= block_pixels;
    float m = block_sum;
    float var = block_sum2 - m * m;
    float s = rsqrtf(var + 1e-5f);
    if (threadIdx.x == 0 && mean != nullptr) {
        mean[blockIdx.x] = m;
    }
    if (threadIdx.x == 0 && rstd != nullptr) {
        rstd[blockIdx.x] = s;
    }

    for (int i = threadIdx.x; i < block_pixels; i += blockDim.x) {
        int c_mod_group = (i / img_size) % group_size;
        float n = s * (x[i] - m);
        out[i] = n * weight[c_mod_group] + bias[c_mod_group];
    }
}

extern "C"
__global__ void groupnorm_backward_kernel(
    const float* dout, const float* x, const float* mean, const float* rstd, const float* weight,
    float* dx, float* dweight, float* dbias,
    int B, int C, int img_size, int group_size, int n_groups
) {
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    __shared__ float shared_sum[32]; // block_size max is 1024 = 32 * 32 warps
    __shared__ float shared_sum2[32]; // warps will be writing into shared memeory after warp-reduce
    int num_warps = blockDim.x / 32;
    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;
    int block_pixels = img_size * group_size;
    // group index
    int g = blockIdx.x % n_groups;

    // move pointers
    dout += blockIdx.x * img_size * group_size;
    x += blockIdx.x * img_size * group_size;
    dx += blockIdx.x * img_size * group_size;
    weight += g * group_size;
    dweight += g * group_size;
    dbias += g * group_size;

    float m_val = mean[blockIdx.x];
    float rstd_val = rstd[blockIdx.x];


    // calculate the two mean terms in the group dimension
    // first is dout * weight, and second is dout * weight * norm
    // where norm = (x - mean) * rstd
    float w_dout_thread = 0.0f;
    float w_dout_norm_thread = 0.0f;
    for (int i = threadIdx.x; i < block_pixels; i += blockDim.x) {
        int c_mod_group = (i / img_size) % group_size;
        float cur_w_dout = weight[c_mod_group] * dout[i];
        w_dout_thread += cur_w_dout;
        float norm = (x[i] - m_val) * rstd_val;
        w_dout_norm_thread += cur_w_dout * norm;
    }
    // warp reduce
    float w_dout_warp = cg::reduce(warp, w_dout_thread, cg::plus<float>{});
    float w_dout_norm_warp = cg::reduce(warp, w_dout_norm_thread, cg::plus<float>{});
    // store warp sum in shared mem
    shared_sum[warp_id] = w_dout_warp;
    shared_sum2[warp_id] = w_dout_norm_warp;
    __syncthreads();

    // load warp sums from shared memory
    w_dout_warp = (lane_id < num_warps) ? shared_sum[lane_id] : 0.0f;
    w_dout_norm_warp = (lane_id < num_warps) ? shared_sum2[lane_id] : 0.0f;
    float w_dout_block = cg::reduce(warp, w_dout_warp, cg::plus<float>{});
    float w_dout_norm_block = cg::reduce(warp, w_dout_norm_warp, cg::plus<float>{});
    w_dout_block /= block_pixels;
    w_dout_norm_block /= block_pixels;

    // update dx
    for (int i = threadIdx.x; i < block_pixels; i += blockDim.x) {
        // in bounds of image
        // accumulate dw and db
        float dout_val = dout[i];
        float norm = (x[i] - m_val) * rstd_val;

        // update dx
        int c_mod_group = (i / img_size) % group_size;
        float w_dout = weight[c_mod_group] * dout_val;
        dx[i] = (w_dout - w_dout_block - norm * w_dout_norm_block) * rstd_val;
    }
    // update dw and db
    // use different methods when the image size is large or small

    // if the image size is larger than the block size
    // loop over the channels and use the whole block on each channel
    // otherwise, assign each warp to a channel
    // in either case image size must be larger than the warp size
    //assert(img_size % warp.size() == 0);
    assert(blockDim.x % warp.size() == 0);
    if (img_size % blockDim.x == 0) {
        for (int c = 0; c < group_size; c++) {
            float dw_thread = 0.0f;
            float db_thread = 0.0f;
            for (int i = threadIdx.x; i < img_size; i += blockDim.x) {
                float dout_val = dout[i];
                db_thread += dout_val;
                float norm = (x[i] - m_val) * rstd_val;
                dw_thread += dout_val * norm;
            }

            // move pointers
            dout += img_size;
            x += img_size;

            // warp reduce
            float dw_warp = cg::reduce(warp, dw_thread, cg::plus<float>{});
            float db_warp = cg::reduce(warp, db_thread, cg::plus<float>{});
            ////// store warp sum in shared mem
            if (lane_id == 0) {
                shared_sum[warp_id] = dw_warp;
                shared_sum2[warp_id] = db_warp;
            }
            __syncthreads();
            // use the first thread to reduce the shared memory sums and save to global memory
            if (threadIdx.x == 0) {
                float dw_block = 0.0f;
                float db_block = 0.0f;
                for (int i = 0; i < num_warps; i++) {
                    dw_block += shared_sum[i];
                    db_block += shared_sum2[i];
                }
                atomicAdd(dweight + c, dw_block);
                atomicAdd(dbias + c, db_block);
            }
        }
    } else {
        // if group size is large, need to loop over the group channels with the whole block
        int block_reps = ceil_div(group_size, num_warps);
        for (int br = 0; br < block_reps; br++) {
            float dw_thread = 0.0f;
            float db_thread = 0.0f;

            int ch = br * num_warps + warp_id;
            if (ch < group_size) {
                const float* dout_ch = dout + ch * img_size;
                const float* x_ch = x + ch * img_size;
                for (int i = lane_id; i < img_size; i += warp.size()) {
                    float dout_val = dout_ch[i];
                    db_thread += dout_val;
                    float norm = (x_ch[i] - m_val) * rstd_val;
                    dw_thread += dout_val * norm;
                }
                
                // warp reduce
                float dw_warp = cg::reduce(warp, dw_thread, cg::plus<float>{});
                float db_warp = cg::reduce(warp, db_thread, cg::plus<float>{});
                // since each warp takes care of an entire image
                // directly store result
                if (lane_id == 0) {
                    atomicAdd(dweight + ch, dw_warp);
                    atomicAdd(dbias + ch, db_warp);
                }
            }
        }
    }
}