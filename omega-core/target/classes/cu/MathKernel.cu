
#include <hip/hip_runtime.h>
#define BLOCK 1024 


extern "C"
__global__ void mean_cov(float* x,float* mean,int number,int channel,int height,int width)
{
    
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < channel; index += blockDim.x * gridDim.x) {
		
		float val = 0;
		
		for(int n = 0;n<number;n++) {	
			for(int h = 0;h<height;h++) {
				for(int w = 0;w<width;w++) {
				
					val += x[n * channel * height * width + index * height * width + h * width + w];
	
				}
			}
		}	
		
		mean[index] = val / (number *  height * width);
	}

}


extern "C"
__global__ void mean_full(float* x,float* mean,int number,int width)
{
    
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < width; index += blockDim.x * gridDim.x) {
		
		float val = 0;
		
		for(int n = 0;n<number;n++) {	
				
			val += x[n * width + index];
	
		}	
		
		mean[index] = val / number;
	}

}

extern "C"
__global__ void var_cov(float* x,float* mean,float* var,int number,int channel,int height,int width)
{
    
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < channel; index += blockDim.x * gridDim.x) {
		
		float val = 0;
		
		float mean_val = mean[index];
		
		for(int n = 0;n<number;n++) {	
			for(int h = 0;h<height;h++) {
				for(int w = 0;w<width;w++) {
					
					float x_val = x[n * channel * height * width + index * height * width + h * width + w];
					
					val += powf((x_val - mean_val), 2);
	
				}
			}
		}	
		
		var[index] = val / (number * height * width - 1);
		//var[index] = val / (number * height * width);
	}

}

extern "C"
__global__ void var_full(float* x,float* mean,float* var,int number,int width)
{
    
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < width; index += blockDim.x * gridDim.x) {
		
		float val = 0;
		
		float mean_val = mean[index];
		
		for(int n = 0;n<number;n++) {	
				
			float x_val = x[n * width + index];
				
			val += powf((x_val - mean_val), 2);
	
		}	
		
		var[index] = val / (number - 1);
		//var[index] = val / number;
	}

}


extern "C"
__global__ void std_fn(float* var,float* std,float eta,int n)
{
    
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < n; index += blockDim.x * gridDim.x) {
		
		std[index] = sqrt(var[index] + eta);
		
	}

}

extern "C"
__global__ void mwa(float* mean,float* var,float* runingMean,float* runingVar,int n,float momentum)
{
    
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < n; index += blockDim.x * gridDim.x) {
		
		runingMean[index] = momentum * runingMean[index] + (1.0f - momentum) * mean[index];
		
		runingVar[index] = momentum * runingVar[index] + (1.0f - momentum) * var[index];
		
	}

}


extern "C"
__global__ void  fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;
            local[id] += (i+id < spatial) ? x[index] : 0;
        }
    }

    __syncthreads();

    if(id == 0){
        mean[filter] = 0;
        for(i = 0; i < threads; ++i){
            mean[filter] += local[i];
        }
        mean[filter] /= spatial * batch;
    }
}


extern "C"
__global__ void  fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;

            local[id] += (i+id < spatial) ? powf((x[index] - mean[filter]), 2) : 0;
        }
    }

    __syncthreads();

    if(id == 0){
        variance[filter] = 0;
        for(i = 0; i < threads; ++i){
            variance[filter] += local[i];
        }
        variance[filter] /= (spatial * batch - 1);
        //variance[filter] /= (spatial * batch);
    }
}