
#include <hip/hip_runtime.h>
#define BLOCK 1024 

extern "C"
__global__ void pooling_forward(int n, int w, int h, int c, float *input, float *output)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    id /= c;
    int b = id;

    int i;
    int out_index = (k + c*b);
    output[out_index] = 0;
    for(i = 0; i < w*h; ++i){
        int in_index = i + h*w*(k + b*c);
        output[out_index] += input[in_index];
    }
    output[out_index] /= w*h;
}

extern "C"
__global__ void pooling_backward(int n, int w, int h, int c, float *out_delta, float *in_delta)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    id /= c;
    int b = id;

    int i;
    int out_index = (k + c*b);
    for(i = 0; i < w*h; ++i){
        int in_index = i + h*w*(k + b*c);
        //in_delta[in_index] += out_delta[out_index] / (w*h);
        in_delta[in_index] = out_delta[out_index] / (w*h);
    }
}
