#include "hip/hip_runtime.h"
#define BLOCK 1024 
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

extern "C"
__global__ void layernorm_forward_kernel(float* __restrict__ out, float* __restrict__ mean, float* __restrict__ rstd,
                                    const float*  __restrict__ inp, const float*  __restrict__ weight,
                                    const float* __restrict__ bias, int N, int C) {
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    __shared__ float shared_sum[32]; // block_size max is 1024 = 32 * 32 warps
    __shared__ float shared_sum2[32]; // warps will be writing into shared memeory after warp-reduce
    int num_warps = blockDim.x / 32;
    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;
    int idx = blockIdx.x; // simpoy one block per row
    // the row of input that this group of threads is responsible for
    const float* x = inp + idx * C;
    // thread coarsening through the row, reduce the sum in series
    float thread_sum = 0.0; // stores sum(x)
    float thread_sum2 = 0.0; // stores sum(x**2)
    // for (int i = C + threadIdx.x - blockDim.x; i >= 0; i -= blockDim.x) {
    for (int i = threadIdx.x; i < C; i += blockDim.x) {
        float xi = x[i];
        thread_sum += xi;
        thread_sum2 += xi * xi;
    }
    // warp-level reduction
    float warp_sum = cg::reduce(warp, thread_sum, cg::plus<float>{}); // sum(x)
    float warp_sum2 = cg::reduce(warp, thread_sum2, cg::plus<float>{}); // sum(x**2)
    // store the warp-level reduction in shared memory (we could have lane_id == 0 guard but not needed)
    shared_sum[warp_id] = warp_sum;
    shared_sum2[warp_id] = warp_sum2;
    __syncthreads();
    // load results from shared memory to threads, pad with zeros for threads that are out of bounds
    warp_sum = (lane_id < num_warps) ? shared_sum[lane_id] : 0.0f;
    warp_sum2 = (lane_id < num_warps) ? shared_sum2[lane_id] : 0.0f;
    // now reduce the warp-level reductions
    float block_sum = cg::reduce(warp, warp_sum, cg::plus<float>{}); // sum(x)
    float block_sum2 = cg::reduce(warp, warp_sum2, cg::plus<float>{}); // sum(x**2)
    // mean, var, rstd
    block_sum /= C; // mean(x)
    block_sum2 /= C; // mean(x**2)
    float m = block_sum;
    float var = block_sum2 - m * m;
    float s = rsqrtf(var + 1e-5f);
    // store the mean, no need to cache it
    if(threadIdx.x == 0 && mean != nullptr) {
        __stcs(mean + idx, m);
    }
    // store the rstd, no need to cache it
    if(threadIdx.x == 0 && rstd != nullptr) {
        __stcs(rstd + idx, s);
    }
    // final normalization and scaling by weight/bias
    float* o = out + idx * C;
    for (int i = threadIdx.x; i < C; i += blockDim.x) {
        float n = s * (__ldcs(x+i) - m);
        __stcs(o+i, n * weight[i] + bias[i]);
    }
}

__device__ float warpReduceSum(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_xor_sync(0xFFFFFFFF, val, offset);
    }
    return val;
}

__device__ void atomicAddX(float* addr, float val) {
    atomicAdd(addr, val);
}

extern "C"
__global__ void layernorm_backward_kernel(float* dinp, float* dweight, float* dbias, float* scratch,
                        const float* dout, const float* inp, const float* weight, const float* mean, const float* rstd,
                        int B, int T, int C) {
    extern __shared__ float shared[]; // size = 2 * C + 1
    int warpId = threadIdx.x / warpSize; // warp index within a block
    int warpsInBlock = blockDim.x / warpSize;
    int base_idx = blockIdx.x * warpsInBlock + warpId;
    int warpThreadIdx = threadIdx.x % warpSize; // Thread index within the warp
    int warps_in_grid = gridDim.x * warpsInBlock;

    // the first half of shared memory is bias, second is weight
    float* dbias_shared = shared;
    float* dweight_shared = shared + C;

    // init shared memory to zero
    #pragma unroll 4
    for(int i = threadIdx.x; i < C; i+= blockDim.x){
       dbias_shared[i] = 0.0f;
       dweight_shared[i] = 0.0f;
    }
    int *tmp_flag = (int*)(shared + C*2);
    __syncthreads();

    for (int idx = base_idx; idx < B * T; idx += warps_in_grid) {
        int b = idx / T;
        int t = idx % T;

        const float* dout_bt = dout + b * T * C + t * C;
        const float* inp_bt = inp + b * T * C + t * C;
        float* dinp_bt = dinp + b * T * C + t * C;
        const float mean_bt = (float)mean[b * T + t];
        const float rstd_bt = (float)rstd[b * T + t];

        // first: two reduce operations
        float dnorm_mean = 0.0f;
        float dnorm_norm_mean = 0.0f;
        for (int i = warpThreadIdx; i < C; i  += warpSize) {
            float norm_bti = ((float)inp_bt[i] - mean_bt) * rstd_bt;
            float dnorm_i = (float)weight[i] * (float)dout_bt[i];
            dnorm_mean += dnorm_i;
            dnorm_norm_mean += dnorm_i * norm_bti;
        }
        dnorm_mean = warpReduceSum(dnorm_mean);
        dnorm_norm_mean = warpReduceSum(dnorm_norm_mean);

        dnorm_mean = dnorm_mean / C;
        dnorm_norm_mean = dnorm_norm_mean / C;

        // now iterate again and accumulate all the gradients
        for (int i = warpThreadIdx; i < C; i += warpSize) {
            float dout_i = (float)__ldcs(&dout_bt[i]);
            float norm_bti = ((float)__ldcs(&inp_bt[i]) - mean_bt) * rstd_bt;
            float dnorm_i = (float)weight[i] * dout_i;
            // gradient contribution to bias
            atomicAdd(&dbias_shared[i], dout_i);
            // gradient contribution to weight
            atomicAdd(&dweight_shared[i], norm_bti * dout_i);
            // gradient contribution to input
            float dval = 0.0f;
            dval += dnorm_i; // term 1
            dval -= dnorm_mean; // term 2
            dval -= norm_bti * dnorm_norm_mean; // term 3
            dval *= rstd_bt; // final scale
            dinp_bt[i] = dinp_bt[i] + dval;
        }
    }

    // Accumulate into a FP32 scratchpad
    // BF16 atomics are potentially much slower... and this is more precise!
    __syncthreads();
    float* scratch_dbias = scratch;
    float* scratch_dweight = scratch + C;
    int* scratchFlag = (int*)(scratch + (2 * C));
    for(int i = threadIdx.x; i < C; i+= blockDim.x) {
        atomicAdd(&scratch_dbias[i], dbias_shared[i]);
        atomicAdd(&scratch_dweight[i], dweight_shared[i]);
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        *tmp_flag = atomicAdd(scratchFlag, 1);
    }
    __syncthreads();
    if (*tmp_flag == gridDim.x-1) {
        for(int i = threadIdx.x; i < C; i+= blockDim.x) {
            // todo - potentially do stochastic rounding here as well
            dbias[i] = (float)scratch_dbias[i];
            dweight[i] = (float)scratch_dweight[i];
        }
    }
}

extern "C"
__global__ void layernorm_forward_kernel5(float* __restrict__ out, float* __restrict__ mean, float* __restrict__ rstd,
                                    const float*  __restrict__ inp, const float*  __restrict__ weight,
                                    const float* __restrict__ bias, int N, int C) {
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    __shared__ float shared_sum[32]; // block_size max is 1024 = 32 * 32 warps
    __shared__ float shared_sum2[32]; // warps will be writing into shared memeory after warp-reduce
    int num_warps = blockDim.x / 32;
    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;
    int idx = blockIdx.x; // simpoy one block per row
    // the row of input that this group of threads is responsible for
    const float* x = inp + idx * C;
    // thread coarsening through the row, reduce the sum in series
    float thread_sum = 0.0; // stores sum(x)
    float thread_sum2 = 0.0; // stores sum(x**2)
    // for (int i = C + threadIdx.x - blockDim.x; i >= 0; i -= blockDim.x) {
    for (int i = threadIdx.x; i < C; i += blockDim.x) {
        float xi = x[i];
        thread_sum += xi;
        thread_sum2 += xi * xi;
    }
    // warp-level reduction
    float warp_sum = cg::reduce(warp, thread_sum, cg::plus<float>{}); // sum(x)
    float warp_sum2 = cg::reduce(warp, thread_sum2, cg::plus<float>{}); // sum(x**2)
    // store the warp-level reduction in shared memory (we could have lane_id == 0 guard but not needed)
    shared_sum[warp_id] = warp_sum;
    shared_sum2[warp_id] = warp_sum2;
    __syncthreads();
    // load results from shared memory to threads, pad with zeros for threads that are out of bounds
    warp_sum = (lane_id < num_warps) ? shared_sum[lane_id] : 0.0f;
    warp_sum2 = (lane_id < num_warps) ? shared_sum2[lane_id] : 0.0f;
    // now reduce the warp-level reductions
    float block_sum = cg::reduce(warp, warp_sum, cg::plus<float>{}); // sum(x)
    float block_sum2 = cg::reduce(warp, warp_sum2, cg::plus<float>{}); // sum(x**2)
    // mean, var, rstd
    block_sum /= C; // mean(x)
    block_sum2 /= C; // mean(x**2)
    float m = block_sum;
    float var = block_sum2 - m * m;
    float s = rsqrtf(var + 1e-5f);
    // store the mean, no need to cache it
    if(threadIdx.x == 0 && mean != nullptr) {
        __stcs(mean + idx, m);
    }
    // store the rstd, no need to cache it
    if(threadIdx.x == 0 && rstd != nullptr) {
        __stcs(rstd + idx, s);
    }
    // final normalization and scaling by weight/bias
    float* o = out + idx * C;
    for (int i = threadIdx.x; i < C; i += blockDim.x) {
        float n = s * (__ldcs(x+i) - m);
        __stcs(o+i, n * weight[i] + bias[i]);
    }
}

extern "C"
__global__ void layernorm_backward_kernel3(float* dinp, float* dweight, float* dbias,
                        const float* dout, const float* inp, const float* weight, const float* mean, const float* rstd,
                        int B, int T, int C) {
    extern __shared__ float shared[]; // size = 2 * C

    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int base_idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();

    // the first half of shared memory is bias, second is weight
    float* dbias_shared = shared;
    float* dweight_shared = shared + C;

    // init shared memory to zero
    #pragma unroll 4
    for(int i = threadIdx.x; i < C; i+= blockDim.x){
       dbias_shared[i] = 0.0f;
       dweight_shared[i] = 0.0f;
    }
    __syncthreads();

    int warps_in_grid = gridDim.x * warp.meta_group_size();
    for (int idx = base_idx; idx < B * T; idx += warps_in_grid) {
        int b = idx / T;
        int t = idx % T;

        const float* dout_bt = dout + b * T * C + t * C;
        const float* inp_bt = inp + b * T * C + t * C;
        float* dinp_bt = dinp + b * T * C + t * C;
        const float mean_bt = (float)mean[b * T + t];
        const float rstd_bt = (float)rstd[b * T + t];

        // first: two reduce operations
        float dnorm_mean = 0.0f;
        float dnorm_norm_mean = 0.0f;
        for (int i = warp.thread_rank(); i < C; i  += warp.size()) {
            float norm_bti = ((float)inp_bt[i] - mean_bt) * rstd_bt;
            float dnorm_i = (float)weight[i] * (float)dout_bt[i];
            dnorm_mean += dnorm_i;
            dnorm_norm_mean += dnorm_i * norm_bti;
        }
        dnorm_mean = cg::reduce(warp, dnorm_mean, cg::plus<float>{});
        dnorm_norm_mean = cg::reduce(warp, dnorm_norm_mean, cg::plus<float>{});
        dnorm_mean = dnorm_mean / C;
        dnorm_norm_mean = dnorm_norm_mean / C;

        // now iterate again and accumulate all the gradients
        for (int i = warp.thread_rank(); i < C; i += warp.size()) {
            float dout_i = (float)__ldcs(&dout_bt[i]);
            float norm_bti = ((float)__ldcs(&inp_bt[i]) - mean_bt) * rstd_bt;
            float dnorm_i = (float)weight[i] * dout_i;
            // gradient contribution to bias
            atomicAdd(&dbias_shared[i], dout_i);
            // gradient contribution to weight
            atomicAdd(&dweight_shared[i], norm_bti * dout_i);
            // gradient contribution to input
            float dval = 0.0f;
            dval += dnorm_i; // term 1
            dval -= dnorm_mean; // term 2
            dval -= norm_bti * dnorm_norm_mean; // term 3
            dval *= rstd_bt; // final scale
            dinp_bt[i] = (float)((float)dinp_bt[i] + dval);
        }
    }
    __syncthreads();

    for(int i = threadIdx.x; i < C; i+= blockDim.x) {
        atomicAddX(&dbias[i], (float)dbias_shared[i]);
        atomicAddX(&dweight[i], (float)dweight_shared[i]);
    }
}

extern "C"
__global__ void layernorm_backward_kernel1(float* dinp, float* dweight, float* dbias,
                        const float* dout, const float* inp, const float* weight, const float* mean, const float* rstd,
                        int B, int T, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= B*T) return;
    int b = idx / T;
    int t = idx % T;

    const float* dout_bt = dout + b * T * C + t * C;
    const float* inp_bt = inp + b * T * C + t * C;
    float* dinp_bt = dinp + b * T * C + t * C;
    const float mean_bt = mean[b * T + t];
    const float rstd_bt = rstd[b * T + t];

    // first: two reduce operations
    float dnorm_mean = 0.0f;
    float dnorm_norm_mean = 0.0f;
    for (int i = 0; i < C; i++) {
        float norm_bti = (inp_bt[i] - mean_bt) * rstd_bt;
        float dnorm_i = weight[i] * dout_bt[i];
        dnorm_mean += dnorm_i;
        dnorm_norm_mean += dnorm_i * norm_bti;
    }
    dnorm_mean = dnorm_mean / C;
    dnorm_norm_mean = dnorm_norm_mean / C;

    // now iterate again and accumulate all the gradients
    for (int i = 0; i < C; i++) {
        float norm_bti = (inp_bt[i] - mean_bt) * rstd_bt;
        float dnorm_i = weight[i] * dout_bt[i];
        // gradient contribution to bias
        atomicAdd(&dbias[i], dout_bt[i]);
        // gradient contribution to weight
        atomicAdd(&dweight[i], norm_bti * dout_bt[i]);
        // gradient contribution to input
        float dval = 0.0f;
        dval += dnorm_i; // term 1
        dval -= dnorm_mean; // term 2
        dval -= norm_bti * dnorm_norm_mean; // term 3
        dval *= rstd_bt; // final scale
        dinp_bt[i] += dval;
    }
}

extern "C"
__global__ void layernorm_backward_kernel7(float* dinp, float* dweight, float* dbias, float* scratch,
                        const float* dout, const float* inp, const float* weight, const float* mean, const float* rstd,
                        int B, int T, int C) {
    extern __shared__ float shared[]; // size = 2 * C + 1
    int warpId = threadIdx.x / warpSize; // warp index within a block
    int warpsInBlock = blockDim.x / warpSize;
    int base_idx = blockIdx.x * warpsInBlock + warpId;
    int warpThreadIdx = threadIdx.x % warpSize; // Thread index within the warp
    int warps_in_grid = gridDim.x * warpsInBlock;

    // the first half of shared memory is bias, second is weight
    float* dbias_shared = shared;
    float* dweight_shared = shared + C;

    // init shared memory to zero
    #pragma unroll 4
    for(int i = threadIdx.x; i < C; i+= blockDim.x){
       dbias_shared[i] = 0.0f;
       dweight_shared[i] = 0.0f;
    }
    int *tmp_flag = (int*)(shared + C*2);
    __syncthreads();

    for (int idx = base_idx; idx < B * T; idx += warps_in_grid) {
        int b = idx / T;
        int t = idx % T;

        const float* dout_bt = dout + b * T * C + t * C;
        const float* inp_bt = inp + b * T * C + t * C;
        float* dinp_bt = dinp + b * T * C + t * C;
        const float mean_bt = (float)mean[b * T + t];
        const float rstd_bt = (float)rstd[b * T + t];

        // first: two reduce operations
        float dnorm_mean = 0.0f;
        float dnorm_norm_mean = 0.0f;
        for (int i = warpThreadIdx; i < C; i  += warpSize) {
            float norm_bti = ((float)inp_bt[i] - mean_bt) * rstd_bt;
            float dnorm_i = (float)weight[i] * (float)dout_bt[i];
            dnorm_mean += dnorm_i;
            dnorm_norm_mean += dnorm_i * norm_bti;
        }
        dnorm_mean = warpReduceSum(dnorm_mean);
        dnorm_norm_mean = warpReduceSum(dnorm_norm_mean);

        dnorm_mean = dnorm_mean / C;
        dnorm_norm_mean = dnorm_norm_mean / C;

        // now iterate again and accumulate all the gradients
        for (int i = warpThreadIdx; i < C; i += warpSize) {
            float dout_i = (float)__ldcs(&dout_bt[i]);
            float norm_bti = ((float)__ldcs(&inp_bt[i]) - mean_bt) * rstd_bt;
            float dnorm_i = (float)weight[i] * dout_i;
            // gradient contribution to bias
            atomicAdd(&dbias_shared[i], dout_i);
            // gradient contribution to weight
            atomicAdd(&dweight_shared[i], norm_bti * dout_i);
            // gradient contribution to input
            float dval = 0.0f;
            dval += dnorm_i; // term 1
            dval -= dnorm_mean; // term 2
            dval -= norm_bti * dnorm_norm_mean; // term 3
            dval *= rstd_bt; // final scale
            dinp_bt[i] = (float)((float)dinp_bt[i] + dval);
        }
    }

    // Accumulate into a FP32 scratchpad
    // BF16 atomics are potentially much slower... and this is more precise!
    __syncthreads();
    float* scratch_dbias = scratch;
    float* scratch_dweight = scratch + C;
    int* scratchFlag = (int*)(scratch + (2 * C));
    for(int i = threadIdx.x; i < C; i+= blockDim.x) {
        atomicAdd(&scratch_dbias[i], dbias_shared[i]);
        atomicAdd(&scratch_dweight[i], dweight_shared[i]);
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        *tmp_flag = atomicAdd(scratchFlag, 1);
    }
    __syncthreads();
    if (*tmp_flag == gridDim.x-1) {
        for(int i = threadIdx.x; i < C; i+= blockDim.x) {
            // todo - potentially do stochastic rounding here as well
            dbias[i] = (float)scratch_dbias[i];
            dweight[i] = (float)scratch_dweight[i];
        }
    }
}